﻿#include"tree.cuh"
#include<random>
int main()
{
    // searching nSearch items in an array of n elements
    int n = 10000000;
    const int nSearch = 10000000;

    Sloth::Tree<int, int> tree;
    std::unordered_map<int, int> map;
    std::vector<int> key(n);
    std::vector<int> value(n);
    for (int i = 0; i < n; i++)
    {
        key[i] = i;
        value[i] = i;
    }
    unsigned int seed = 0;

    // unique random numbers generated
    std::shuffle(key.begin(), key.end(), std::default_random_engine(seed));

    std::cout << " =========================== Benchmarking Initialization ============================ " << std::endl;
    // build tests
    for (int i = 0; i < 5; i++)
    {
        size_t t;
        std::cout << "------------------------------------------------------------------------------------" << std::endl;
        int valueFound = -1;
        bool found = false;
        for (int i = 0; i < 100; i++)
        {
            {
                Sloth::Bench bench(&t);
                tree.Build(key, value);
            }
            valueFound = -1;
            found |= tree.FindKeyCpu(key[15], valueFound);
            if (valueFound == -1)
            {
                // try again to check if only "find" is failed or "build" is failed
                found = false;
                valueFound = -1;
                found |= tree.FindKeyCpu(key[15], valueFound);
                if (valueFound == -1)
                {
                    std::cout << "Build-failure!" << std::endl;
                    std::cout << " found: " << (found ? "yes" : "no") << std::endl;
                    std::cout << " found value: " << valueFound << " real value: " << value[15] << std::endl;
                    std::cout << "error: tree traversal could not find the key that was inserted." << std::endl;
                    return 0;
                }

                std::cout << "Find-failure!" << std::endl;
                std::cout << " found: " << (found ? "yes" : "no") << std::endl;
                std::cout << " found value: " << valueFound << " real value: " << value[15] << std::endl;
                std::cout << "error: tree traversal could not find the key that was inserted." << std::endl;
                return 0;
            }
            if(i%10 == 0)
                std::cout << "build gpu: " << t / 1000000000.0 << "s" << std::endl;
        }
        std::cout << " found: " << (found ? "yes" : "no") << std::endl;
        std::cout << " found value: " << valueFound << " real value: " << value[15] << std::endl;
        map.clear();
        
        {
            Sloth::Bench bench(&t);
            for (int j = 0; j < n; j++)
                map[key[j]] = value[j];
        }
        std::cout << "build std::unordered_map: " << t / 1000000000.0 << "s" << std::endl;

    }


    std::cout << " =========================== Benchmarking Search Operation ============================ " << std::endl;


    // search tests
    // nSearch consumes too much video-memory. beware.
    // it allocates enough queue-space for nSearch CUDA threads
    // if 1 million keys are to be searched, do it in 100 steps using 10k chunks

    std::vector<int> keys(nSearch);
    std::vector<int> values(nSearch);
    std::vector<char> conditions(nSearch);
    std::vector<int> valuesBruteForce(nSearch);
    std::vector<char> conditionsBruteForce(nSearch);
    for (int i = 0; i < nSearch; i++)
    {        
        keys[i] = i;
    }
    // searching doesn't require unique keys but same code was reused anyway.
    std::shuffle(keys.begin(), keys.end(), std::default_random_engine(seed));

    size_t t;
    for (int i = 0; i < 5; i++)
    {
        {
            Sloth::Bench bench(&t);
            for (int j = 0; j < nSearch; j++)
            {
                bool cond = false;
                int val = -1;
                auto it = map.find(keys[j]);
                if (it != map.end())
                {
                    val = it->second;
                    cond = true;
                }
                conditionsBruteForce[j] = cond;
                valuesBruteForce[j] = val;
            }

        }
        std::cout << "find std::unordered_map: " << t / 1000000000.0 << "s" << std::endl;

        for (int k = 0; k < 100; k++)
        {
            {
                Sloth::Bench bench(&t);
                tree.FindKeyGpu(keys, values, conditions);
            }
            if(k%10==0)
                std::cout << "simple find gpu: " << t / 1000000000.0 << "s" << std::endl;
        }

        //checking error
        for (int i = 0; i < nSearch; i++)
        {
            bool fail = false;
            if (conditionsBruteForce[i] != conditions[i])
            {
                std::cout << "Error: tree-find failed (condition)!" << std::endl;
                fail = true;
            }

            if (valuesBruteForce[i] != values[i])
            {
                std::cout << "Error: tree-find failed (value)!" << std::endl;
                fail = true;
            }
            if (fail)
            {
                std::cout << "tree result: " << values[i] << " brute-force result: " << valuesBruteForce[i] << std::endl;
                std::cout << "tree condition: " << (int)conditions[i] << " brute-force condition: " << (int)conditionsBruteForce[i] << std::endl;
                return 0;
            }
        }
    }

    return 0;
}
